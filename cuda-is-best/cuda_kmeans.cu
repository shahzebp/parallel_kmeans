#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>

#include "kmeans.h"

static int nextPowerOfTwo(int n) {
    int res = 0;
    while(n > 0){
        n >>= 1;
        res = (res<<1) | 1;
    }
    return (res+1);
}

__host__ __device__ static
float euclid_dist_2(int numCoords, int numObjs, int numClusters, float *objects, float *clusters, int objectId, int clusterId){
    float ans=0.0;

    for (int i = 0; i < numCoords; i++) {
        float temp = (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
        ans += pow(temp,2);
    }
    ans = sqrt(ans);
    return ans;
}

__global__ static
void find_nearest_cluster(int numCoords, int numObjs, int numClusters, float *objects, float *deviceClusters, int *membership,
                          int *intermediates){
    extern __shared__ char sharedMemory[];

    unsigned char *membershipChanged = (unsigned char *)sharedMemory;

    membershipChanged[threadIdx.x] = 0;

    int objectId =  threadIdx.x + (blockDim.x * blockIdx.x);

    if (objectId < numObjs) {
        float min_dist;
        int index  = -1;
        min_dist = FLT_MAX;
        float *clusters = deviceClusters;
        for (int i=0; i<numClusters; i++) {
            float dist = euclid_dist_2(numCoords, numObjs, numClusters,
                                 objects, clusters, objectId, i);
            index = (dist < min_dist ? (min_dist = dist, i): index);
        }

        if (membership[objectId] != index) {
            membership[objectId] = index;
            membershipChanged[threadIdx.x] = 1;
        }

        __syncthreads();
        unsigned int s = blockDim.x / 2;
        while(s > 0) {
            membershipChanged[threadIdx.x] += ((threadIdx.x < s) ? membershipChanged[threadIdx.x + s] : 0);
            s >>= 1;
            __syncthreads();
        }
         
        if (!(threadIdx.x)) {
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}

__global__ static
void compute_delta(int *deviceIntermediates, int numIntermediates, int numIntermediates2){
    
    numIntermediates2 >>= 1;
    extern __shared__ unsigned int intermediates[];

    intermediates[threadIdx.x] =
        ((threadIdx.x >= numIntermediates) ? 0 : deviceIntermediates[threadIdx.x]);

    __syncthreads();
    
    unsigned int s =  numIntermediates2;
    while(s > 0) {
        intermediates[threadIdx.x] += ((threadIdx.x < s) ? intermediates[threadIdx.x + s] : 0);
        s >>= 1;
        __syncthreads();
    }

    if (!(threadIdx.x)) {
        deviceIntermediates[0] = intermediates[0];
    }
}

float** cuda_kmeans(float **objects, int numCoords, int numObjs, int numClusters, int *membership){

    float  **dimObjects;
    malloc2D(dimObjects, numCoords, numObjs, float);
    for (int i = 0; i < numCoords; i++) {
        for (int j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    float *deviceClusters;
    float  **dimClusters;
    malloc2D(dimClusters, numCoords, numClusters, float);
    for (int i = 0; i < numCoords; i++) {
        for (int j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }

    memset(membership, -1, numObjs*sizeof(int));

    int *newClusterSize; 
    newClusterSize = (int*) calloc(numClusters, sizeof(int));

    float  **newClusters;
    malloc2D(newClusters, numCoords, numClusters, float);
    memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

    unsigned int numThreadsPerClusterBlock = 128;
    unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;

    unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char);

    unsigned int numReductionThreads =
        nextPowerOfTwo(numClusterBlocks);
    unsigned int reductionBlockSharedDataSize =
        numReductionThreads * sizeof(unsigned int);


    float *deviceObjects;
    int *deviceMembership;
    int *deviceIntermediates;

    hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float));
    hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float));
    hipMalloc(&deviceMembership, numObjs*sizeof(int));
    hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int));

    hipMemcpy(deviceObjects, dimObjects[0], numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceMembership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice);

    for(int loop = 0; loop < 500; loop++){
        hipMemcpy(deviceClusters, dimClusters[0], numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice);

        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize();

        compute_delta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);

        hipDeviceSynchronize();

        int d;
        hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost);
        float delta = (float)d;

        hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost);

        for (int i=0; i<numObjs; i++) {
            newClusterSize[membership[i]] += 1;
            for (int j=0; j<numCoords; j++)
                newClusters[j][membership[i]] += objects[i][j];
        }

        for (int i=0; i<numClusters; i++) {
            for (int j=0; j<numCoords; j++) {
                if (newClusterSize[i] != 0)
                    dimClusters[j][i] = (newClusters[j][i] / (1.0*newClusterSize[i]));
                newClusters[j][i] = 0;
            }
            newClusterSize[i] = 0;
        }

        if(delta > 0.001){
            break;
        }
    }   

    float  **clusters;
    malloc2D(clusters, numClusters, numCoords, float);
    for (int i = 0; i < numClusters; i++) {
        for (int j = 0; j < numCoords; j++) {
            clusters[i][j] = dimClusters[j][i];
        }
    }

    return clusters;
}

